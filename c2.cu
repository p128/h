

#include <hip/hip_runtime.h>
#include <stdio.h>




__global__ void use_local_memory_GPU(float in)
{
    float f;   
    f = in;     
}



__global__ void use_global_memory_GPU(float *array)
{
  
    array[threadIdx.x] = 2.0f * (float) threadIdx.x;
}




__global__ void use_shared_memory_GPU(float *array)
{
    
    int i, index = threadIdx.x;
    float average, sum = 0.0f;

 
    __shared__ float sh_arr[10];

    
    sh_arr[index] = array[index];

    __syncthreads();   
    for (i=0; i<index; i++) { sum += sh_arr[i]; }
    average = sum / (index + 1.0f);

        printf("Thread id = %d\t Average = %f\n",index,average);
    
    if (array[index] > average) { array[index] = average; }

   
    sh_arr[index] = 3.14;
}

int main(int argc, char **argv)
{
  
    use_local_memory_GPU<<<1, 10>>>(2.0f);

   
    float h_arr[10];   
float *d_arr;       

    hipMalloc((void **) &d_arr, sizeof(float) * 10);
    
    hipMemcpy((void *)d_arr, (void *)h_arr, sizeof(float) * 10, hipMemcpyHostToDevice);
    
    use_global_memory_GPU<<<1, 10>>>(d_arr);  
   
    hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 10, hipMemcpyDeviceToHost);
   

  
    use_shared_memory_GPU<<<1, 10>>>(d_arr);
    
    hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 10, hipMemcpyHostToDevice);
  


    hipDeviceSynchronize();
    return 0;
}
